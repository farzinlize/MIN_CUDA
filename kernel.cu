#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
extern "C" {
	#include "helper_functions.h"
}
#include <stdio.h>
#include <hip/hip_runtime.h>

#ifdef SYNCTHREAD
void __syncthreads();
#endif

__device__	int device_min(int a, int b) {
	return a < b ? a : b;
}

__global__ void minKernel(int *a_in, int *out){
	extern __shared__ int a_s[];
	unsigned int tid_block = threadIdx.x;
	unsigned int tid = blockDim.x * blockIdx.x + tid_block;
	
	a_s[tid_block] = a_in[tid];

	for (unsigned int s = blockDim.x/2; s > 0 ; s >>= 1) {
		if (tid_block < s)
			a_s[tid_block] = device_min(a_s[tid_block], a_s[tid_block + s]);
		__syncthreads();
	}

	if (tid_block == 0)
		out[blockIdx.x] = a_s[0];
}

int find_min_seq(int *a, int size) {
	int min = a[0];
	for (int i = 1; i < size; i++) {
		if (a[i] < min)
			min = a[i];
	}
	return min;
}

int main(){
	int size = 1024 * 1024, block_size = 1024;
	int *a_h, *a_d, *out_d, *device_out_h;

	initialize_data_random(&a_h, size);
	initialize_data_zero(&device_out_h, block_size);
	
	int min_seq = find_min_seq(a_h, size);

	CUDA_CHECK_RETURN(hipMalloc((void **)&a_d, sizeof(int)*size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&out_d, sizeof(int)*block_size));

	dim3 grid_dim(1024, 1, 1);
	dim3 block_dim(block_size, 1, 1);

	CUDA_CHECK_RETURN(hipMemcpy(a_d, a_h, sizeof(int)*size, hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

	minKernel <<<grid_dim, block_dim, sizeof(int)*block_size, NULL >>> (a_d, out_d);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipMemcpy(device_out_h, out_d, sizeof(int)*block_size, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
	CUDA_CHECK_RETURN(hipGetLastError());

	for(int k=0; k < block_size ;k++){
		printf("%d\t", device_out_h[k]);
	}

	int min_parralel = find_min_seq(device_out_h, block_size);

	printf("Parallel_min: %d \nSeq_min: %d", min_parralel, min_seq);

    return 0;
}